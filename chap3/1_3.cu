/*
    Ex 3.1: Square Matrix Addition

    v3: each thread works on 1 column

    This is the worst out of the 3 since
    C stores memory in row-major order.
*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

__global__ void matrixAdd(float* A, float* B, float* C, int M) {
    int i = threadIdx.x;
    int offset;
    for (int j=0; j<M; j++) {
        offset = j*M + i;
        C[offset] = A[offset] + B[offset];
    }
}

int main(void) {

    // parameters
    int M = 10;
    int numElements = M * M;
    size_t size = numElements * sizeof(float);

    // allocate host matrices
    float* h_A = (float*) malloc(size);
    float* h_B = (float*) malloc(size);
    float* h_C = (float*) malloc(size);

    // initialize host matrices
    int i, j, offset;
    for (i = 0; i <  M; i++) {
        for (j = 0; j < M; j++) {
            offset = i*M + j;
            h_A[offset] = 1.;
            h_B[offset] = 1.;
        }
    }

    // allocate device matrices
    float* d_A;
    float* d_B;
    float* d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // host matrices -> device matrices
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // kernel launch
    int numThreads = M;
    int numBlocks = 1;
    matrixAdd<<<numBlocks, numThreads>>>(d_A, d_B, d_C, M);

    // device matrices -> host matrices
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // print result
    // should be a matrix of 2's
    for (i = 0; i <  M; i++)
        for (j = 0; j < M; j++)
            printf("%f ", h_C[i*M + j]);

    // free device and host memory
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    free(h_A); free(h_B); free(h_C);

    return 0;
}
