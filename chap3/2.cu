/*
    Ex 2: Matrix Vector Multiplication
*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

__global__ void matrixVecMultiply(float* A, float* B, float* C, int M) {
    int i = threadIdx.x;
    int offset;
    float sum = 0;
    for (int j=0; j<M; j++) {
        offset = i*M + j;
        sum += A[offset] * B[j];
    }
    C[i] = sum;
}

int main(void) {

    // parameters
    int M = 3;
    size_t sizeMatrix = M * M * sizeof(float);
    size_t sizeVec = M * sizeof(float);

    // allocate host matrices
    float* h_A = (float*) malloc(sizeMatrix);
    float* h_B = (float*) malloc(sizeVec);
    float* h_C = (float*) malloc(sizeVec);

    // initialize host matrices
    int i, j, offset;
    float count1, count2 = 0.;
    for (i = 0; i <  M; i++) {
        for (j = 0; j < M; j++) {
            offset = i*M + j;
            h_A[offset] = ++count1;
        }
        h_B[i] = ++count2;
    }

    // print matrices
    printf("A: ");
    for (i = 0; i <  M; i++)
        for (j = 0; j < M; j++)
            printf("%f ", h_A[i*M + j]);
    printf("\nB: ");
    for (i = 0; i <  M; i++)
            printf("%f ", h_B[i]);

    // allocate device matrices
    float* d_A;
    float* d_B;
    float* d_C;
    hipMalloc((void**)&d_A, sizeMatrix);
    hipMalloc((void**)&d_B, sizeVec);
    hipMalloc((void**)&d_C, sizeVec);

    // host matrices -> device matrices
    hipMemcpy(d_A, h_A, sizeMatrix, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeVec, hipMemcpyHostToDevice);

    // kernel launch
    int numThreads = M;
    int numBlocks = 1;
    matrixVecMultiply<<<numBlocks, numThreads>>>(d_A, d_B, d_C, M);

    // device matrices -> host matrices
    hipMemcpy(h_C, d_C, sizeVec, hipMemcpyDeviceToHost);

    // print result
    printf("\nC: ");
    for (i = 0; i <  M; i++)
        printf("%f ", h_C[i]);
    printf("\n");

    // free device and host memory
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    free(h_A); free(h_B); free(h_C);

    return 0;
}
