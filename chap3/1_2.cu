/*
    Ex 3.1: Square Matrix Addition

    v2: each thread works on 1 row
*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

__global__ void matrixAdd(float* A, float* B, float* C, int M) {
    int i = threadIdx.x;
    int offset;
    for (int j=0; j<M; j++) {
        offset = i*M + j;
        C[offset] = A[offset] + B[offset];
    }
}

int main(void) {

    // parameters
    int M = 10;
    int numElements = M * M;
    size_t size = numElements * sizeof(float);

    // allocate host matrices
    float* h_A = (float*) malloc(size);
    float* h_B = (float*) malloc(size);
    float* h_C = (float*) malloc(size);

    // initialize host matrices
    int i, j, offset;
    for (i = 0; i <  M; i++) {
        for (j = 0; j < M; j++) {
            offset = i*M + j;
            h_A[offset] = 1.;
            h_B[offset] = 1.;
        }
    }

    // allocate device matrices
    float* d_A;
    float* d_B;
    float* d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // host matrices -> device matrices
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // kernel launch
    int numThreads = M;
    int numBlocks = 1;
    matrixAdd<<<numBlocks, numThreads>>>(d_A, d_B, d_C, M);

    // device matrices -> host matrices
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // print result
    for (i = 0; i <  M; i++)
        for (j = 0; j < M; j++)
            printf("%f ", h_C[i*M + j]);

    // free device and host memory
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    free(h_A); free(h_B); free(h_C);

    return 0;
}
